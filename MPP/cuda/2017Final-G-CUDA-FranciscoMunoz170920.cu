
/*
  CPP_CONTEST=2017
  CPP_PROBLEM=I
  CPP_LANG=CUDA
  CPP_PROCESSES_PER_NODE=saturno 1
*/



/* RECORD
Francisco Muñoz García
September 20, 2017
in CESGA
time 1520
speed-up 9.80
*/




#include <hip/hip_runtime.h>
#include <stdlib.h>

__device__ int count(int ld,int n,char *a,char *b) //Each CUDA thread do this work and is called from kernel so we change to __device__
{
  int i,j;
  int value=0;
  for(i=0;i < n;i++)
    for(j=0;j < n;j++)
      if(a[i*ld+j]==b[i*n+j])
        value++;
  return value;
}

/*
We create one thread for each element in matrix sizexsize. Each element compare its matrix and save the results in a matrix. For that reason
each thread has an associated element in the matrix.
*/
__global__ void mask(char* a, char* b, int* temp, int n, int m) {
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int j = blockIdx.y*blockDim.y + threadIdx.y;
	int size = n-m;
	if((i<size) && (j<size)) {
		temp[i*size+j]=count(n,m,&a[i*n+j],b);
	}
}

int  sec(int n,char *a,int m,char *b)
{
  int i, j;
  int maximum=0,value;
  int size = n-m;
  int nbytes_a = sizeof(char)*n*n;
  int nbytes_b = sizeof(char)*m*m;
  int nBytes_temp = sizeof(int)*size*size;

  int* temp =(int*) malloc(sizeof(int)*size*size); 
  int* temp_d;
  char* a_d;
  char* b_d;

  int bl_dim1 = 4;
  int bl_dim2 = 8;

  dim3 block(bl_dim1,bl_dim2);

  //we need n-m threads

  int gsx = size / bl_dim1;
  if(size%bl_dim1) gsx++;
  int gsy = size / bl_dim2;
  if(size%bl_dim2) gsy++;
  dim3 grid(gsx, gsy);


  //We reserve memory for GPU
  hipMalloc((void **) &temp_d, nBytes_temp);
  hipMalloc((void**) &a_d, nbytes_a);
  hipMalloc((void**) &b_d, nbytes_b);
  
  //Transfers here
  hipMemset(temp_d, 0, nBytes_temp*sizeof(char)); //All the values should stat with zeros because each thread add values from that initial zero.

  hipMemcpy(a_d, a, nbytes_a, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b, nbytes_b, hipMemcpyHostToDevice);



  //call the kernel
  mask<<<grid, block>>>(a_d, b_d, temp_d, n,m );


  //We transfer the results to RAM
  hipMemcpy(temp, temp_d, nBytes_temp, hipMemcpyDeviceToHost);

  hipFree((void**)temp_d);
  hipFree((void**)a_d);
  hipFree((void**)b_d);

  //Once we have the results for each comparition we only have to know which is the best. We do this in sequencial mode.
  maximum = temp[0];
  for(int i=1; i<size*size;i++) {
	if(temp[i]>maximum)
		maximum=temp[i];
  }

  free(temp);
  return maximum;
}


