
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>
#include <signal.h>
#include <unistd.h>

//Spanish Parallel programming Contest 2017. Problem I - heterosolar 
//Maximum coincidence with a mask in 2D. CUDA version.
//Schema for In/Out, validation and execution time

void generar(char *m, int t,int sup) {
  int i;

  for (i = 0; i < t; i++) {
      m[i] = (char) (((1. * rand()) / RAND_MAX)*sup)+'a';
  }
}

void escribir(char *m, int t) {
  int i;

  for (i = 0; i < t; i++) {
      printf("%c ", m[i]);
  }
  printf("\n");
}

/*
c
c     mseconds - returns elapsed milliseconds since Jan 1st, 1970.
c
*/
long long mseconds(){
  struct timeval t;
  gettimeofday(&t, NULL);
  return t.tv_sec*1000 + t.tv_usec/1000;
}

static void alarm_handler(int sig) {
  fprintf(stderr, "Time Limit Exceeded\n");
  abort();
}

extern int sec(int,char *,int,char *);

int main(int argc,char *argv[]) {
  int N,M,cuantos;
  bool correcto=true;
  int semilla,upper;
  char *A,*B;
  long long ti,tf,tt=0;

  //FILE *stats_file = fopen("stats", "w");

  struct sigaction sact;
  sigemptyset(&sact.sa_mask);
  sact.sa_flags = 0;
  sact.sa_handler = alarm_handler;
  sigaction(SIGALRM, &sact, NULL);
  alarm(40);  /* time limit */

  scanf("%d",&cuantos);

  for(int i=0;i < cuantos;i++)
  {
      scanf("%d",&N);                   // Matrices size
      scanf("%d",&M);                   // mask size
      scanf("%d",&semilla);             // seed for random generation
      scanf("%d",&upper);                 // upper value for random generation

// Space for the matrix, the values, rows and columns
      A = (char *) malloc(sizeof(double)*N*N);
      B = (char *) malloc(sizeof(double)*M*M);

      srand(semilla);
     
      generar(A,N*N,upper);
      generar(B,M*M,upper);
/*#ifdef DEBUG
    escribir(A,N*N);
    escribir(B,M*M);
#endif*/
    ti=mseconds(); 
    printf("%d\n",sec(N,A,M,B));
    tf=mseconds(); 
      if(i!=0) tt+=tf-ti;

      free(A);
      free(B);
  }
  
    // fprintf(stats_file, "%Ld\n", tt);
    // fclose(stats_file);
  printf("%Ld\n", tt);
  return 0;
}